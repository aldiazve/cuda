#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

// Max Threads per block in GeForce 210
#define TxB 1024

__global__
void rgba_to_grey_kernel(const uchar4* const originalImage,
                       unsigned char* const outImage,
                       int numRows, int numCols, int totalThreads)
{

  int initIteration, endIteration;

  int id = (blockDim.x * blockIdx.x) + threadIdx.x;

  initIteration = (720*480/totalThreads) * id;

  if (id == totalThreads - 1)
    endIteration = 720*480;
  else
    endIteration = initIteration + ((720*480 / totalThreads));

  int index = 0;

  for (int aux = initIteration; aux < endIteration; aux++){
    int j = aux % 720;
    int i = (aux - j) / 720;
    index = aux * 3;
    int x = j * (numCols/720.0);
    int y = i * (numRows/480.0);
  
    int indexAux = (x + y * numCols);
    uchar4 px = originalImage[indexAux]; // thread pixel to process
    outImage[index + 2] = px.x; 
    outImage[index + 1] = px.y; 
    outImage[index] = px.z;
    
  }
}

void rgba_to_grey(uchar4 * const d_rgbaImage,
                  unsigned char* const d_greyImage, size_t numRows, size_t numCols, int aBlockSize, int aGridSize)
{

  // Since it does not matter the relative position of a pixel
  // the block - grid assign strategy will simply be to cover
  // all pixels secuencially in 'x' axis
  //cols => 852
  //rows => 480
  int totalThreads = aBlockSize * aGridSize;
  printf("Total threads %d", totalThreads);
  // long long int total_px = 852*480;  // total pixels
  // long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(aBlockSize, 1, 1);
  const dim3 gridSize(aGridSize, 1, 1);
  rgba_to_grey_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols, totalThreads);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}